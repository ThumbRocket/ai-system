#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <string>
#include <math.h>
#include <assert.h>

#define BLOCK_SIZE 16

using namespace std;


__global__ void Convolution(float* A, float* B, float* C, int numARows, int numACols, int numBRows, int numBCols, int numCRows, int numCCols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    float sum;
   
    //A * B = C
    if(row<numCRows&&col<numCCols){
        sum =0;
        for(size_t i=0;i<numBCols ;i++){
            
            for(size_t j=0;j<numBRows;j++){
                sum+=A[row*numACols+i*numACols+col+j]*B[i*numBCols+j];
            }
    
        }
        C[row*numCCols+col]=sum;
    
    }    

}

__host__ void cpu_Convolution(float *A, float *B, float *C, int asize, int bsize){

	int csize=asize-bsize+1;
	int sum;
	for(int i=0;i<csize;i++){
		
		for (int j=0;j<csize;j++){
			sum=0;
			for(int k=0;k<bsize;k++){
				
				for(int l=0;l<bsize;l++){
					sum+=A[i*asize+k*asize+j+l]*B[k*bsize+l];

				}
				
			}
			C[i*csize+j]=sum;
		}



	}
	




}

void randomInit(float* data, int size)
{
	for (int i = 0; i < size; ++i)
		data[i] = rand() %10;
}

int main()
{
	srand(2006);
	int a, b,c;
	hipEvent_t start_G, stop_G;
	float gpu_miliseconds, cpu_miliseconds;
	hipEventCreate(&start_G);
	hipEventCreate(&stop_G);
	printf("Please type in the size of input and filter ( type in '5 3'-> 5 x 5 matrix and 3 x 3 filter) \n");
    scanf("%d %d", &a, &b);
	c=a-b+1;
	unsigned int size_A = a * a;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);

	unsigned int size_B = b * b;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);

	unsigned int size_C = c * c;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C = (float*)malloc(mem_size_C);
	float* h_C_cpu = (float*)malloc(mem_size_C);
	randomInit(h_A, size_A);
	for (int i = 0; i < size_B; ++i)
    {
        h_B[i] = rand() %4;
    }

	float* d_A;
	float* d_B;
	float* d_C;
    //for (int i = 0; i < size_A; ++i)
    //{
    //    h_A[i] = i;
    //}
    
	hipMalloc((void**)&d_A, mem_size_A);
	hipMalloc((void**)&d_B, mem_size_B);
	hipMalloc((void**)&d_C, mem_size_C);

	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
	
	unsigned int grid_rows= (c+BLOCK_SIZE-1) / BLOCK_SIZE;
	unsigned int grid_cols= (c+BLOCK_SIZE-1) / BLOCK_SIZE;
	
	dim3 dimGrid(grid_cols,grid_rows);	
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	
	hipEventRecord(start_G,0);
	Convolution << < dimGrid, dimBlock >> >(d_A, d_B, d_C, a, a, b, b, c, c);



	
	hipDeviceSynchronize();

	

	hipEventRecord(stop_G,0);

	hipEventSynchronize(stop_G);

	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	
	hipEventElapsedTime(&gpu_miliseconds, start_G, stop_G);

	printf("\nTime took to compute matrix A(%d x %d) with filter B(%d x %x) on GPU is %f ms  \n \n", a, a,b,b, gpu_miliseconds);
    printf("matrix A\n");
	for (int i = 0;i < a;i++)
	{
		for (int j = 0;j < a;j++)
		{
			printf("%f\t", h_A[i*a + j]);
		}
		printf("\n");
	}printf("\n");
    printf("matrix B\n");
    for (int i = 0;i < b;i++)
	{
		for (int j = 0;j < b;j++)
		{
			printf("%f\t", h_B[i*b + j]);
		}
		printf("\n");
	}printf("\n");
    printf("matrix C\n");
	for (int i = 0;i < c;i++)
	{
		for (int j = 0;j < c;j++)
		{
			printf("%f\t", h_C[i*c + j]);
		}
		printf("\n");
	}
	hipEventRecord(start_G, 0);
	cpu_Convolution(h_A, h_B, h_C_cpu, a,b);
	hipEventRecord(stop_G,0);
	hipEventSynchronize(stop_G);
	hipEventElapsedTime(&cpu_miliseconds, start_G, stop_G);
	printf("\nTime took to compute matrix A(%d x %d) with filter B(%d x %x) on CPU is %f ms  \n \n", a, a,b,b, cpu_miliseconds);
    
	for (int i = 0;i < c;i++)
	{
		for (int j = 0;j < c;j++)
		{
			printf("%f\t", h_C_cpu[i*c + j]);
		}
		printf("\n");
	}

	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_cpu);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return EXIT_SUCCESS;
}

